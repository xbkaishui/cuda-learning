#include <iostream>
#include <hip/hip_runtime.h>


__global__ void gpuAdd(int d_a, int d_b, int *d_c)
{
    *d_c = d_a + d_b;
}

// We will use a convention in this book that host variables will be prefixed with h_ and device variables will be prefixed with d_. This is not compulsory; it is just done so that readers can understand the concepts easily without any confusion between host and device.
int main(void)
{
    // Defining host variable to store answer
    int h_c;
    // Defining device pointer
    int *d_c;
    // Allocating memory for device pointer
    hipMalloc((void **)&d_c, sizeof(int));
    // Kernel call by passing 1 and 4 as inputs and storing answer in d_c
    //<< <1,1> >> means 1 block is executed with 1 thread per block
    gpuAdd<<<1, 1>>>(1, 4, d_c);
    // Copy result from device memory to host memory
    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("1 + 4 = %d\n", h_c);
    // Free up memory
    hipFree(d_c);
    return 0;
}